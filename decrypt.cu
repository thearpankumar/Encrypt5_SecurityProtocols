#include <fstream>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

extern "C" {
    void decrypt_files(const char* encrypted_file, const char* key_file, const char* decrypted_file);
}

long long int read_file_to_memmory(FILE *pInfile, int *pPointer) {
    if(pInfile != NULL) {
        int mIndex = 0;
        int mSize = fread(pPointer + mIndex, 1, sizeof(int), pInfile);
        long long int mFileSize = 0;
        while(mSize != 0) {
            mFileSize = mFileSize + mSize;
            ++mIndex;
            mSize = fread(pPointer + mIndex, 1, mSize, pInfile);
        }
        return mFileSize;
    }
    return 0;
}

long long int write_file_from_memmory(FILE *pOutFile, int *pPointer, long long int pFileSize) {
    if(pOutFile != NULL) {
        pFileSize = fwrite(pPointer, 1, pFileSize, pOutFile);
        return pFileSize;
    }
    return 0;
}

__global__ void generate_decrypted(int *pDataPointer, int *pRandomData, int *pEncryptedData, long long int pSize) {
    long long int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index <= (pSize / sizeof(int))) {
        (*(pEncryptedData + index)) = (*(pDataPointer + index)) ^ (*(pRandomData + index));
    }
}

void decrypt_files(const char* encrypted_file, const char* key_file, const char* decrypted_file) {
    FILE *inFile;
    FILE *outFile;
    FILE *keyFile;
    inFile = fopen(encrypted_file, "rb");
    keyFile = fopen(key_file, "rb");
    outFile = fopen(decrypted_file, "wb");

    if (!inFile) {
        std::cerr << "Error: Could not open encrypted file " << encrypted_file << std::endl;
        return;
    }
    if (!keyFile) {
        std::cerr << "Error: Could not open key file " << key_file << std::endl;
        return;
    }
    if(!outFile) {
        std::cerr << "Error: Could not open output file " << decrypted_file << std::endl;
        return;
    }

    int *encryptedDataPointer = new int[268435456];
    long long int fileSize = read_file_to_memmory(inFile, encryptedDataPointer);
    int *keyDataPointer = new int[fileSize / sizeof(int) + 100];
    int *decryptedDataPointer = new int[fileSize / sizeof(int) + 100];
    fileSize = read_file_to_memmory(keyFile, keyDataPointer);

    int *d_encryptedDataPointer;
    hipMalloc((void**)&d_encryptedDataPointer, fileSize);
    int *d_keyPointer;
    hipMalloc((void**)&d_keyPointer, fileSize);
    int *d_decryptedDataPointer;
    hipMalloc((void**)&d_decryptedDataPointer, fileSize);

    hipMemcpy(d_encryptedDataPointer, encryptedDataPointer, fileSize, hipMemcpyHostToDevice);
    hipMemcpy(d_keyPointer, keyDataPointer, fileSize, hipMemcpyHostToDevice);

    generate_decrypted<<<fileSize / 64 + 1, 64>>>(d_encryptedDataPointer, d_keyPointer, d_decryptedDataPointer, fileSize);

    hipMemcpy(decryptedDataPointer, d_decryptedDataPointer, fileSize, hipMemcpyDeviceToHost);
    fileSize = write_file_from_memmory(outFile, decryptedDataPointer, fileSize);

    fclose(inFile);
    fclose(outFile);
    fclose(keyFile);
}